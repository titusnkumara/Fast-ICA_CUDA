#include "hip/hip_runtime.h"
// 
 
#include <stdio.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <Eigen/Dense>
#include <Eigen/Eigenvalues>


#include <fcntl.h>    /* For O_RDWR */
#include <unistd.h>   /* For open(), creat() */
#include <stdlib.h>
#include <cmath>
#include <iostream>
#include <fstream>

#include <sys/time.h>
#include <time.h>

#include <string.h>
#include <assert.h>

#include <cula_lapack.h>

#include <cula_lapack_device.h>
#include <hipblas.h>
#include "lib/helpers.cuh"

#define imin(X, Y)  ((X) < (Y) ? (X) : (Y))
//void checkStatus(culaStatus status);

using namespace Eigen;
using namespace std;
using Eigen::ArrayXXd ;
using Eigen::MatrixXd ;

#include "common.h"
#include <hipblas.h>
#include "lib/myTimer.h"


#include <hip/hip_runtime.h>
#include <hipsolver.h>

hipblasHandle_t handle;

__global__ 
void applyCube(float *original,float * computed,float * cubeDerivation, int n,int p) 
{
	
	//converted to 2d
	
	int y = blockIdx.y*blockDim.y+threadIdx.y; //n
	int x = blockIdx.x*blockDim.x+threadIdx.x; //p
	
	if(x<p && y<n){
		float originalFloat = original[y*p+x];
		float tmp = originalFloat*originalFloat;
		cubeDerivation[y*p+x]=3*tmp;
		computed[y*p+x] =tmp*originalFloat;
	}
	
}



__global__ 
void findMean(float * computed,float * alternative,int n,int p) 
{
	
	int x = blockIdx.x*blockDim.x+threadIdx.x;
	
	computed[x]=alternative[x]/p;
	
}


__global__ 
void multiplyColumnVise(float *original,float * calculated,float * factor,int n,int p) 
{
	
	//int x = blockIdx.x*blockDim.x+threadIdx.x;
	int y = blockIdx.y*blockDim.y+threadIdx.y; //n
	int x = blockIdx.x*blockDim.x+threadIdx.x; //p
	//int i;
	
	if(y<n && x<p){
		float mulFactor;
		int index;
		
		mulFactor = factor[y];
		index = x*n+y;
		calculated[index] = original[index]*mulFactor;

	}	
}


__global__ 
void subtractMatrices(float * target,float *A,float * B,int n) 
{
	
	int x = blockIdx.x*blockDim.x+threadIdx.x;
	if(x<n*n){
		target[x] = A[x]-B[x];
	}
		
}



/*
BLAS functions
*/


void create_blas_handler(){
	
	// Create a handle for CUBLAS
	hipblasCreate(&handle);
	
}

void destroy_blas_handler(){
	
	 // Destroy the handle
	hipblasDestroy(handle);
	
}


// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul(float *A, float *B, float *C, const int m, const int k, const int n) {
	int lda=m,ldb=k,ldc=m;
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;
	
	// Do the actual multiplication
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
	hipDeviceSynchronize();
	
}


// Improved version for transpose multiplication
void gpu_blas_mmulImprove(float *A, float *B, float *C, const int m, const int k, const int n,const float p_) {
	int lda=m,ldb=k,ldc=m;
	const float alf = 1/p_;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;
	// Do the actual multiplication
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
	hipDeviceSynchronize();
}

__global__
void memSetInCuda(float *d_singleArray,float num,int sizeofSingleArray){
	
	int x = blockIdx.x*blockDim.x+threadIdx.x;
	if(x<sizeofSingleArray){
		
		d_singleArray[x] = num;
	}
	
}

int runSVDonCUDA(MatrixXd& input,VectorXd& singularValue,MatrixXd& singularVectors,int ROWS,int COLS){
	//initilaize matrix
	//int ROWS =1000;
	//int COLS =3;
	MatrixXf DataMatrix = input.cast<float>();
	
    float* h_A = DataMatrix.data();
	
	
	int M = ROWS;
    int N = COLS;
	
	culaStatus status;
	
	/* Setup SVD Parameters */
    int LDA = M;
    int LDU = M;
    int LDVT = N;
   
    float* A = NULL;
    float* S = NULL;
    float* U = NULL;
    float* VT = NULL;
	
	time_t begin_time;
    time_t end_time;
    int cula_time;

    char jobu = 'N';
    char jobvt = 'A';
	
			//cout<<"checking h_A 1 "<<h_A[0]<<" "<<h_A[1]<<" "<<h_A[1]<<endl;
	
	hipMalloc((void**)&A, M*N*sizeof(float ));checkCudaError();
			//cout<<"checking h_A 2 "<<h_A[0]<<" "<<h_A[1]<<" "<<h_A[1]<<endl;
	hipMemcpy( A, h_A,  M*N*sizeof(float ), hipMemcpyHostToDevice );checkCudaError();
			//cout<<"checking h_A 3 "<<h_A[0]<<" "<<h_A[1]<<" "<<h_A[1]<<endl;
			//float * test = (float *)malloc(M*N*sizeof(float ));
			//hipMemcpy( test, A,  M*N*sizeof(float ), hipMemcpyDeviceToHost );checkCudaError();
			//cout<<"checking h_A 4 "<<h_A[0]<<" "<<h_A[1]<<" "<<h_A[1]<<endl;
			
			//checking elements
			//cout<<test[0]<<" "<<test[1]<<" "<<test[2]<<endl;
	//sleep(1);
	//A = (float*)malloc(M*N*sizeof(float));
	hipMalloc( (void**)&S, imin(M,N)*sizeof(float));checkCudaError();
    //S = (float*)malloc(imin(M,N)*sizeof(float));
    //U = (float*)malloc(N*N*sizeof(float));
	hipMalloc( (void**)&VT, LDVT*N*sizeof(float));checkCudaError();
    //VT = (float*)malloc(LDVT*N*sizeof(float));

	/*
	if(!A || !S  || !VT) 
    {
        free(A);
        //free(U);
        free(S);
        free(VT);

        return EXIT_FAILURE;
    }
	*/
	/* Initialize CULA */
    status = culaInitialize();
    //checkStatus(status);
	 /* Perform singular value decomposition CULA */
    printf("Performing singular value decomposition using CULA ... \n");

    time(&begin_time);
    status = culaDeviceSgesvd(jobu, jobvt, M, N, A, LDA, S, U, LDU, VT, LDVT);
    //checkStatus(status);
    time(&end_time);

	
    cula_time = (int)difftime( end_time, begin_time);
	
	
	
	culaShutdown();
	//copy back data,maybe should move before culashutdown
	float* S_tmp;
	float * VT_tmp;
	S_tmp = (float*)malloc(imin(M,N)*sizeof(float));
	VT_tmp = (float*)malloc(LDVT*N*sizeof(float));
	hipMemcpy(S_tmp,S,imin(M,N)*sizeof(float),hipMemcpyDeviceToHost);checkCudaError();
	hipMemcpy(VT_tmp,VT,LDVT*N*sizeof(float),hipMemcpyDeviceToHost);checkCudaError();
	
	
	for (int i = 0; i < singularVectors.size(); i++){
		
		*(singularVectors.data() + i) = (double)VT_tmp[i];
	}
	
	for (int i = 0; i < singularValue.size(); i++){
		
		*(singularValue.data() + i) = (double)S_tmp[i];
	}
	
	hipDeviceSynchronize();checkCudaError();

    return EXIT_SUCCESS;

}
/*
preprocessVariables initializeCudaForPreprocess(MatrixXd& S,preprocessVariables preprocessData,int n,int p){
	//This is transform so row columns interchanged
	timestamp_t prepr1 = get_timestamp();
	
	MatrixXf DataMatrix = S.cast<float>();
	
    preprocessData.h_A = DataMatrix.data();
	
	cout<<"preprocessData.h_A"<<preprocessData.h_A[0]<<" "<<preprocessData.h_A[1]<<" "<<preprocessData.h_A[1]<<endl;

	timestamp_t prepr2 = get_timestamp();
	cout<<"initializeCudaForPreprocess "<<(prepr2 - prepr1) / 1000000.0L<<endl;
	return preprocessData;
}
*/


cudaVar initializeCuda(MatrixXd& W,MatrixXd& X1,MatrixXd& w_init,cudaVar cudaVariables,int n,int p){
	
	//matrix sizes
	
	MatrixXf f_W = W.cast<float>();
	MatrixXf f_X1 = X1.cast<float>();
	MatrixXf f_w_init = w_init.cast<float>();
	
	MatrixXf f_X1Transpose = f_X1.transpose();
	
	const int matsizeX1 = n*p*sizeof(float);
	const int matsizeX1Transpose = p*n*sizeof(float);
	const int matsizeW = n*n*sizeof(float);
	const int matsizeW1 = n*n*sizeof(float);
	
	const int matsizew_init = n*n*sizeof(float);
	const int matsizeProduct = n*p*sizeof(float);
	const int matsizegwtx = n*p*sizeof(float);
	const int matsizeCubeDerivation = n*p*sizeof(float);
	const int matsizeg_wtx = n*1*sizeof(float);
	const int matsizeGwtxIntoXtranspose = n*n*sizeof(float);
	const int matsizeGwtx_into_W = n*n*sizeof(float);
	const int matsizew_init_w_init_T = n*n*sizeof(float);
	const int matsizeEigenValues = n*1*sizeof(float);
	const int matsizeEigenVectors = n*n*sizeof(float);
	const int matsizeEigenRowWise = n*n*sizeof(float);
	
	const int matsizeW1intoWT = n*n*sizeof(float);
	
	const int matsizebw = n*1*sizeof(float);
	const int matsizezw = n*1*sizeof(float);
	
	const int matsizediagonal = n*1*sizeof(float);
	
	const int matsizeit_num = sizeof(int);
	const int matsizerot_num = sizeof(int);
	const int matsizeAnswer = sizeof(float);
	const int matsize_tmp_w_init = n*n*sizeof(float);

	
	//malloc
	hipMalloc( (void**)&cudaVariables.X1, matsizeX1 );
	hipMalloc( (void**)&cudaVariables.X1Transpose, matsizeX1Transpose );
	hipMalloc( (void**)&cudaVariables.W, matsizeW );
	hipMalloc( (void**)&cudaVariables.W1, matsizeW1 );
	hipMalloc( (void**)&cudaVariables.w_init, matsizew_init );
	hipMalloc( (void**)&cudaVariables.product, matsizeProduct );
	hipMalloc( (void**)&cudaVariables.gwtx, matsizegwtx );
	hipMalloc( (void**)&cudaVariables.cubeD, matsizeCubeDerivation );
	hipMalloc( (void**)&cudaVariables.g_wtx, matsizeg_wtx );
	hipMalloc( (void**)&cudaVariables.g_wtx_X1_transpose, matsizeGwtxIntoXtranspose );
	hipMalloc( (void**)&cudaVariables.gwtx_into_W, matsizeGwtx_into_W );
	hipMalloc( (void**)&cudaVariables.w_init_w_init_T, matsizew_init_w_init_T );
	hipMalloc( (void**)&cudaVariables.eigenValues, matsizeEigenValues );
	hipMalloc( (void**)&cudaVariables.eigenVectors, matsizeEigenVectors );
	hipMalloc( (void**)&cudaVariables.eigenRowWise, matsizeEigenRowWise );
	
	hipMalloc( (void**)&cudaVariables.W1intoWT, matsizeW1intoWT );
	hipMalloc( (void**)&cudaVariables.diagonal, matsizediagonal );
	hipMalloc( (void**)&cudaVariables.answer, matsizeAnswer );
	hipMalloc( (void**)&cudaVariables.tmp_w_init, matsize_tmp_w_init );
	
	
	//malloc
	hipMalloc( (void**)&cudaVariables.bw, matsizebw );
	hipMalloc( (void**)&cudaVariables.zw, matsizezw );
	hipMalloc( (void**)&cudaVariables.it_num, matsizeit_num );
	hipMalloc( (void**)&cudaVariables.rot_num, matsizerot_num );
	
	const int sizeofSingleArray = p*1*sizeof(float);
	const int sizeofComputedArray = n*1*sizeof(float);
	hipMalloc( (void**)&cudaVariables.d_singleArray, sizeofSingleArray );
	hipMalloc( (void**)&cudaVariables.d_computeArray, sizeofComputedArray );
	
	hipHostMalloc((void **) &cudaVariables.hostpointer,matsizeW);
	
	
	
	int blockSize = 512;
    int gridSize = (int)ceil(((float)(p))/blockSize);
	//MemSet in CUDA
	memSetInCuda<<<gridSize, blockSize>>>(cudaVariables.d_singleArray,1.0,p*1*sizeof(float));
	//hipMemset(&(cudaVariables.d_singleArray),1.0,p*1*sizeof(float));
	
	//pointers to data
	float *dataFromX1 = f_X1.data();
	float *datafromX1transpose = f_X1Transpose.data();
	float *dataFromW = f_W.data();
	float *dataFromW_init = f_w_init.data();
	
	
	
	//copy data to CUDA
	hipMemcpy( cudaVariables.X1, dataFromX1, matsizeX1, hipMemcpyHostToDevice );
	hipMemcpy( cudaVariables.X1Transpose, datafromX1transpose, matsizeX1Transpose, hipMemcpyHostToDevice );
	hipMemcpy( cudaVariables.W, dataFromW, matsizeW, hipMemcpyHostToDevice );
	hipMemcpy( cudaVariables.w_init, dataFromW_init, matsizew_init, hipMemcpyHostToDevice );


	hipDeviceSynchronize();
	
	return cudaVariables;
	
	
	}


void copyBackW_initfromCUDA(MatrixXd& w_init,MatrixXf& tmp,float * from,float * hostpointer,float * tmp_w_init,int n){

	//copy to tmp_w_init in cuda memory
	hipMemcpy(tmp_w_init,from,sizeof(float)*n*n,hipMemcpyDeviceToDevice);
	//copy to host pointer 
	hipMemcpy(hostpointer,tmp_w_init,n * n * sizeof(float),hipMemcpyDeviceToHost);
	
	for (int i = 0; i < w_init.size(); i++){
		
		*(w_init.data() + i) = (double)hostpointer[i];
	}

	//cout<<"winit"<<endl;
	//cout<<w_init<<endl;

	}

void saveW1inGPU(MatrixXd& W1,cudaVar cudaVariables,int n){
	
	MatrixXf f_W1 = W1.cast<float>();
	float *dataFromW1 = f_W1.data();
	const int matsizeW1 = n*n*sizeof(float);
	
	hipMemcpy( cudaVariables.W, dataFromW1, matsizeW1, hipMemcpyHostToDevice );
	hipDeviceSynchronize();
	
}




void cubeOnGPU(cudaVar cudaVariables,int n,int p){
	float * original=cudaVariables.product;
	float * computed=cudaVariables.gwtx;
	float * cubeDerivation=cudaVariables.cubeD;
	float * g_wtx = cudaVariables.g_wtx;
	
	
	
    dim3 blockSize(16,16);
	dim3 gridSize((int)ceil(((float)p)/blockSize.x),(int)ceil(((float)n)/blockSize.y));
	
    applyCube<<<gridSize, blockSize>>>(original,computed,cubeDerivation,n,p);
	hipDeviceSynchronize();

	float * d_singleArray = cudaVariables.d_singleArray;
	float * d_computeArray = cudaVariables.d_computeArray;

	gpu_blas_mmul(cubeDerivation, d_singleArray, d_computeArray,n,p,1);
	
	blockSize = n;
    gridSize = 1;
	//original mean function
    
    findMean<<<gridSize, blockSize>>>(g_wtx,d_computeArray,n,p);
    hipDeviceSynchronize();
		
	}

	
	
void multiplyColumnViseOnGPU(cudaVar cudaVariables,int n,int p){
	float * original=cudaVariables.W;
	float * calculated = cudaVariables.gwtx_into_W;
	float * factor=cudaVariables.g_wtx;
	
	
		
	dim3 blockSize(16,16);
	dim3 gridSize((int)ceil(((float)p)/blockSize.x),(int)ceil(((float)n)/blockSize.y));
    
    multiplyColumnVise<<<gridSize, blockSize>>>(original,calculated,factor,n,n);
    hipDeviceSynchronize();
    
	}
	
void subtractOnGPU(cudaVar cudaVariables,int n){
	float * gwtx_into_x1transpose_p=cudaVariables.g_wtx_X1_transpose;
	float * gwtx_into_W=cudaVariables.gwtx_into_W;
	float * target=cudaVariables.w_init;

	int blockSize, gridSize;
    blockSize = 512;
    gridSize = (int)ceil(((float)(n*n))/blockSize);

    subtractMatrices<<<gridSize, blockSize>>>(target,gwtx_into_x1transpose_p,gwtx_into_W,n);
    
    hipDeviceSynchronize();

	}

	
void matrixMultiplyonGPU(float * d_A, float * d_B, float * d_C,int n,int p){
	
	gpu_blas_mmul(d_A, d_B, d_C, n, n, p);
	hipDeviceSynchronize();

	}


void matrixMultiplyTransposeImprovedonGPU(cudaVar cudaVariables,float p_,int n,int p){
	float * d_A = cudaVariables.gwtx;
	float * d_B = cudaVariables.X1Transpose;
	float * d_C = cudaVariables.g_wtx_X1_transpose;
	
	gpu_blas_mmulImprove(d_A, d_B, d_C, n, p, n,p_);
	hipDeviceSynchronize();
	
	}
	

